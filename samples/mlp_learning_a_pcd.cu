/*
 * Copyright (c) 2020-2022, NVIDIA CORPORATION.  All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without modification, are permitted
 * provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright notice, this list of
 *       conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright notice, this list of
 *       conditions and the following disclaimer in the documentation and/or other materials
 *       provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the names of its contributors may be used
 *       to endorse or promote products derived from this software without specific prior written
 *       permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND ANY EXPRESS OR
 * IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND
 * FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,
 * BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS;
 * OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT,
 * STRICT LIABILITY, OR TOR (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */
/*
 */

/** @file   mlp-learning-a-pcd.cu
 *  @author Luca De Luigi, CVLab (Unibo)
 *  @brief  Sample application that uses the tiny cuda nn framework to learn a
            3D function that represents a point cloud.
 */

#include <tiny-cuda-nn/common_device.h>
#include <tiny-cuda-nn/config.h>
#include "cnpy/cnpy.h"

#include <chrono>
#include <cstdlib>
#include <fstream>
#include <iostream>
#include <random>
#include <stdexcept>
#include <string>
#include <thread>
#include <vector>

using namespace tcnn;
using precision_t = network_precision_t;

GPUMemory<float> load_coords(const std::string &filename, int &num_points)
{
    cnpy::NpyArray coords_npy = cnpy::npy_load(filename.c_str());
    num_points = coords_npy.shape[0];
    float *coords = coords_npy.data<float>();

    GPUMemory<float> result(num_points * 3);
    result.copy_from_host(coords);

    return result;
}

GPUMemory<float> load_labels(const std::string &filename, int &num_points)
{
    cnpy::NpyArray labels_npy = cnpy::npy_load(filename.c_str());
    num_points = labels_npy.shape[0];
    float *labels = labels_npy.data<float>();

    GPUMemory<float> result(num_points);
    result.copy_from_host(labels);

    return result;
}

int main(int argc, char *argv[])
{
    try
    {
        uint32_t compute_capability = cuda_compute_capability();
        if (compute_capability < MIN_GPU_ARCH)
        {
            std::cerr
                << "Warning: Insufficient compute capability " << compute_capability << " detected. "
                << "This program was compiled for >=" << MIN_GPU_ARCH << " and may thus behave unexpectedly." << std::endl;
        }

        if (argc < 3)
        {
            std::cout << "USAGE: " << argv[0] << " "
                      << "path-to-coords.npy path-to-labels.npy [path-to-optional-config.json]" << std::endl;
            return 0;
        }

        json config = {
            {"loss", {{"otype", "L2"}}},
            {"optimizer", {
                              {"otype", "Adam"},
                              {"learning_rate", 1e-4},
                              {"beta1", 0.9f},
                              {"beta2", 0.99f},
                              {"l2_reg", 0.0f},
                          }},
            {"encoding", {
                             {"otype", "OneBlob"},
                             {"n_bins", 32},
                         }},
            {"network", {
                            {"otype", "FullyFusedMLP"},
                            {"n_neurons", 64},
                            {"n_hidden_layers", 4},
                            {"activation", "ReLU"},
                            {"output_activation", "None"},
                        }},
        };

        if (argc >= 4)
        {
            std::cout << "Loading custom json config '" << argv[3] << "'." << std::endl;
            std::ifstream f{argv[3]};
            config = json::parse(f, nullptr, true, /*skip_comments=*/true);
        }

        // First step: load 3D coordinates and UDF values
        int num_points;
        GPUMemory<float> coords = load_coords(argv[1], num_points);
        GPUMemory<float> labels = load_labels(argv[2], num_points);

        // Second step: create a cuda texture out of coords and labels.
        // It'll be used to generate training data efficiently on the fly.
        hipResourceDesc resDesc;
        memset(&resDesc, 0, sizeof(resDesc));
        resDesc.resType = hipResourceTypeArray;
        resDesc.res.pitch2D.devPtr = image.data();
        resDesc.res.pitch2D.desc = hipCreateChannelDesc(32, 32, 32, 32, hipChannelFormatKindFloat);
        resDesc.res.pitch2D.width = width;
        resDesc.res.pitch2D.height = height;
        resDesc.res.pitch2D.pitchInBytes = width * 4 * sizeof(float);

        hipTextureDesc texDesc;
        memset(&texDesc, 0, sizeof(texDesc));
        texDesc.filterMode = hipFilterModeLinear;
        texDesc.normalizedCoords = true;
        texDesc.addressMode[0] = hipAddressModeClamp;
        texDesc.addressMode[1] = hipAddressModeClamp;
        texDesc.addressMode[2] = hipAddressModeClamp;

        hipResourceViewDesc viewDesc;
        memset(&viewDesc, 0, sizeof(viewDesc));
        viewDesc.format = hipResViewFormatFloat4;
        viewDesc.width = width;
        viewDesc.height = height;

        hipTextureObject_t texture;
        CUDA_CHECK_THROW(hipCreateTextureObject(&texture, &resDesc, &texDesc, &viewDesc));

        // uint32_t n_coords = sampling_width * sampling_height;
        // uint32_t n_coords_padded = (n_coords + 255) / 256 * 256;

        // GPUMemory<float> sampled_image(n_coords * 3);
        // GPUMemory<float> xs_and_ys(n_coords_padded * 2);

        // std::vector<float> host_xs_and_ys(n_coords * 2);
        // for (int y = 0; y < sampling_height; ++y)
        // {
        //     for (int x = 0; x < sampling_width; ++x)
        //     {
        //         int idx = (y * sampling_width + x) * 2;
        //         host_xs_and_ys[idx + 0] = (float)(x + 0.5) / (float)sampling_width;
        //         host_xs_and_ys[idx + 1] = (float)(y + 0.5) / (float)sampling_height;
        //     }
        // }

        // xs_and_ys.copy_from_host(host_xs_and_ys.data());

        // linear_kernel(eval_image<3>, 0, nullptr, n_coords, texture, xs_and_ys.data(), sampled_image.data());

        // save_image(sampled_image.data(), sampling_width, sampling_height, 3, 3, "reference.exr");

        // // Fourth step: train the model by sampling the above image and optimizing an error metric

        // // Various constants for the network and optimization
        // const uint32_t batch_size = 1 << 16;
        // const uint32_t n_training_steps = argc >= 4 ? atoi(argv[3]) : 10000000;
        // const uint32_t n_input_dims = 2;  // 2-D image coordinate
        // const uint32_t n_output_dims = 3; // RGB color

        // hipStream_t inference_stream;
        // CUDA_CHECK_THROW(hipStreamCreate(&inference_stream));
        // hipStream_t training_stream = inference_stream;

        // default_rng_t rng{1337};

        // // Auxiliary matrices for training
        // GPUMatrix<float> training_target(n_output_dims, batch_size);
        // GPUMatrix<float> training_batch(n_input_dims, batch_size);

        // // Auxiliary matrices for evaluation
        // GPUMatrix<float> prediction(n_output_dims, n_coords_padded);
        // GPUMatrix<float> inference_batch(xs_and_ys.data(), n_input_dims, n_coords_padded);

        // json encoding_opts = config.value("encoding", json::object());
        // json loss_opts = config.value("loss", json::object());
        // json optimizer_opts = config.value("optimizer", json::object());
        // json network_opts = config.value("network", json::object());

        // std::shared_ptr<Loss<precision_t>> loss{create_loss<precision_t>(loss_opts)};
        // std::shared_ptr<Optimizer<precision_t>> optimizer{create_optimizer<precision_t>(optimizer_opts)};
        // std::shared_ptr<NetworkWithInputEncoding<precision_t>> network = std::make_shared<NetworkWithInputEncoding<precision_t>>(n_input_dims, n_output_dims, encoding_opts, network_opts);

        // auto trainer = std::make_shared<Trainer<float, precision_t, precision_t>>(network, optimizer, loss);

        // std::chrono::steady_clock::time_point begin = std::chrono::steady_clock::now();

        // float tmp_loss = 0;
        // uint32_t tmp_loss_counter = 0;

        // std::cout << "Beginning optimization with " << n_training_steps << " training steps." << std::endl;

        // for (uint32_t i = 0; i < n_training_steps; ++i)
        // {
        //     bool print_loss = i % 1000 == 0;
        //     bool visualize_learned_func = argc < 5 && i % 1000 == 0;

        //     // Compute reference values at random coordinates
        //     {
        //         generate_random_uniform<float>(training_stream, rng, batch_size * n_input_dims, training_batch.data());
        //         linear_kernel(eval_image<n_output_dims>, 0, training_stream, batch_size, texture, training_batch.data(), training_target.data());
        //     }

        //     // Training step
        //     float loss_value;
        //     {
        //         trainer->training_step(training_stream, training_batch, training_target, &loss_value);
        //     }
        //     tmp_loss += loss_value;
        //     ++tmp_loss_counter;

        //     // Debug outputs
        //     {
        //         if (print_loss)
        //         {
        //             std::chrono::steady_clock::time_point end = std::chrono::steady_clock::now();
        //             std::cout << "Step#" << i << ": "
        //                       << "loss=" << tmp_loss / (float)tmp_loss_counter << " time=" << std::chrono::duration_cast<std::chrono::microseconds>(end - begin).count() << "[µs]" << std::endl;

        //             tmp_loss = 0;
        //             tmp_loss_counter = 0;
        //         }

        //         if (visualize_learned_func)
        //         {
        //             network->inference(inference_stream, inference_batch, prediction);
        //             save_image(prediction.data(), sampling_width, sampling_height, 3, n_output_dims, std::to_string(i) + ".exr");
        //         }

        //         // Don't count visualizing as part of timing
        //         // (assumes visualize_learned_pdf is only true when print_loss is true)
        //         if (print_loss)
        //         {
        //             begin = std::chrono::steady_clock::now();
        //         }
        //     }
        // }
    }
    catch (std::exception &e)
    {
        std::cout << "Uncaught exception: " << e.what() << std::endl;
    }

    return EXIT_SUCCESS;
}
